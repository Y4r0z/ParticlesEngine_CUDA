#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""
#include "scene.cuh"
#include <iostream>
#include <stdio.h>
#include <SFML/Graphics.hpp>
#include <chrono>
#include <string>

__device__ void pAtomicCpy(Particle& p1, Particle& p2)
{


	atomicExch(&p1.m_radius, p2.m_radius);

	atomicExch(&p1.m_curPos.x, p2.m_curPos.x);
	atomicExch(&p1.m_curPos.y, p2.m_curPos.y);

	atomicExch(&p1.m_prevPos.x, p2.m_prevPos.x);
	atomicExch(&p1.m_prevPos.y, p2.m_prevPos.y);
	
	atomicExch(&p1.m_pressure, p2.m_pressure);
	atomicExch(&p1.m_pressureCoef, p2.m_pressureCoef);

	//atomicExch(&p1.m_color.r, p2.m_color.r);
	//atomicExch(&p1.m_color.g, p2.m_color.g);
	//atomicExch(&p1.m_color.b, p2.m_color.b);

	atomicExch(&p1.m_acceleration.x, p2.m_acceleration.x);
	atomicExch(&p1.m_acceleration.y, p2.m_acceleration.y);

}

__device__ bool pCmp(Particle& p1, Particle& p2)
{
	return p1.m_curPos.x == p2.m_curPos.x && p1.m_curPos.y == p2.m_curPos.y &&
		p1.m_prevPos.x == p2.m_prevPos.x && p1.m_prevPos.y == p2.m_prevPos.y &&
		p1.m_radius == p2.m_radius;
}
__device__ void pCpy(Particle& p1, Particle& other)
{
	p1.m_curPos = other.m_curPos;
	p1.m_prevPos = other.m_prevPos;
	p1.m_radius = other.m_radius;
	p1.m_pressure = other.m_pressure;
	p1.m_pressureCoef = other.m_pressureCoef;
	p1.m_color = other.m_color;
	p1.m_acceleration = other.m_acceleration;
	p1.returnColorPressure = other.returnColorPressure;
}

__device__ void kernelCalculatePos(Particle& p, const float dt)
{
	const float vel_x = p.m_curPos.x - p.m_prevPos.x, vel_y = p.m_curPos.y - p.m_prevPos.y;
	p.m_prevPos.x = p.m_curPos.x;
	p.m_prevPos.y = p.m_curPos.y;
	const float m = sqrtf(vel_x * vel_x + vel_y * vel_y);
	const float mc = p.m_pressure * p.m_pressure;
	const float pc = 1.f / (1.f + mc);

	p.m_curPos.x += (vel_x + (p.m_acceleration.x * dt * dt)) * pc;
	p.m_curPos.y += (vel_y + (p.m_acceleration.y * dt * dt)) * pc;

	p.m_pressureCoef = pc;
	p.m_pressure = 0.f;
}

__device__ void kernelAccelerate(Particle& p, sf::Vector2f a)
{
	p.m_acceleration.x += a.x;
	p.m_acceleration.y += a.y;
}

__device__ void kernelApplyConstraint(Particle& p, sf::Vector2f pos1, sf::Vector2f pos2)
{
	const float
		x = p.m_curPos.x,
		y = p.m_curPos.y,
		r = p.m_radius;
	if (x + r > pos2.x)
		p.m_curPos.x = pos2.x - r;
	if (y + r > pos2.y)
		p.m_curPos.y = pos2.y - r;
	if (x - r < pos1.x)
		p.m_curPos.x = pos1.x + r;
	if (y - r < pos1.y)
		p.m_curPos.y = pos1.y + r;
}

__device__ void kernelCollide(Particle& p1, Particle& p2)
{
	const float ax = p1.m_curPos.x - p2.m_curPos.x;
	const float ay = p1.m_curPos.y - p2.m_curPos.y;
	const float ndist = (ax * ax) + (ay * ay);
	const float r2 = p1.m_radius + p2.m_radius;

	if (ndist > r2 * r2)
		return;
	const float dist = sqrtf(ndist); 
	if (!dist)
		return;
	const float delta = ((r2 - dist)) * 0.5f;
	const float nx = ax / dist * delta;
	const float ny = ay / dist * delta;;
	p1.m_curPos.x += nx;
	p1.m_curPos.y += ny;
	p2.m_curPos.x -= nx;
	p2.m_curPos.y -= ny;
	p1.m_pressure += delta / 2.f;
	p2.m_pressure += delta / 2.f;
	
}



__device__ void staticGridCollide(cui pos1, cui pos2, Particle* grid, int* cellCount, cui gridWidth, cui gridHeight, cui cellSize)
{
	if (pos2 < 0 || pos2 >= gridWidth * gridHeight)
		return;

	for (int i{}; i < cellCount[pos1]; ++i)
		for (int j{}; j < cellCount[pos2]; ++j)
		{
			kernelCollide(grid[pos1 * cellSize + i], grid[pos2 * cellSize + j]);
		}
	
}




__global__ void calculateCollisions(Particle* grid, int* cellCount, cui gridWidth, cui gridHeight, cui cellSize, float radius, int* cellCountOut)
{
	const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= gridHeight * gridWidth) return;
	for (int j{}; j < cellCount[i]; ++j)
	{
		// ������� ������� � ������� grid
		const int pos = i * cellSize + j;
		// �������� ����������
		int x = (int)(grid[pos].m_curPos.x / (radius * 2.f));
		int y = (int)(grid[pos].m_curPos.y / (radius * 2.f));
		// �������� ���������
		if (x < 0)
			x = 0;
		if (y < 0)
			y = 0;
		if (x >= gridWidth)
			x = gridWidth - 1;
		if (y >= gridHeight)
			y = gridHeight - 1;
		// �������� �������
		int pos2 = x * gridHeight + y;

		if (pos2 != i && cellCount[pos2] < cellSize)
		{
			// TODO : BUG. ������� ��������
			pCpy(grid[pos2 * cellSize + cellCount[pos2]], grid[pos]);
			pCpy(grid[pos], grid[i * cellSize + (cellCount[i] - 1 )]);
			//atomicAdd(&cellCountOut[pos2], 1);
			//atomicSub(&cellCountOut[i], 1);
			cellCount[pos2]++;
			cellCount[i]--;
		}
	}
}


__global__ void applyGravityKernel(Particle* grid, int* cellCount, cui gridWidth, cui gridHeight, cui cellSize, sf::Vector2f g)
{
	const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= gridHeight * gridWidth) return;
	for (int j{}; j < cellCount[i]; ++j)
		kernelAccelerate(grid[i * cellSize + j], g);
}
__global__ void applyCollisionsKernel(Particle* grid, int* cellCount, cui gridWidth, cui gridHeight, cui cellSize, float radius)
{
	const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= gridHeight * gridWidth) return;
	for (int j{}; j < cellCount[i]; ++j)
	{
		staticGridCollide(i, i, grid, cellCount, gridWidth, gridHeight, cellSize);
		staticGridCollide(i, i + 1, grid, cellCount, gridWidth, gridHeight, cellSize);
		staticGridCollide(i, i - 1, grid, cellCount, gridWidth, gridHeight, cellSize);
		staticGridCollide(i, i + gridHeight, grid, cellCount, gridWidth, gridHeight, cellSize);
		staticGridCollide(i, i + gridHeight + 1, grid, cellCount, gridWidth, gridHeight, cellSize);
		staticGridCollide(i, i + gridHeight - 1, grid, cellCount, gridWidth, gridHeight, cellSize);
		staticGridCollide(i, i - gridHeight, grid, cellCount, gridWidth, gridHeight, cellSize);
		staticGridCollide(i, i - gridHeight + 1, grid, cellCount, gridWidth, gridHeight, cellSize);
		staticGridCollide(i, i - gridHeight - 1, grid, cellCount, gridWidth, gridHeight, cellSize);
	}
}
__global__ void applyConstraintsKernel(Particle* grid, int* cellCount, cui gridWidth, cui gridHeight, cui cellSize, sf::Vector2f b1, sf::Vector2f b2)
{
	const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= gridHeight * gridWidth) return;
	for (int j{}; j < cellCount[i]; ++j)
		kernelApplyConstraint(grid[i * cellSize + j], b1, b2);
}

__global__ void calculatePositionsKernel(Particle* grid, int* cellCount, cui gridWidth, cui gridHeight, cui cellSize, float dt)
{
	const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= gridHeight * gridWidth) return;
	for (int j{}; j < cellCount[i]; ++j)
		kernelCalculatePos(grid[i * cellSize + j], dt);
}



KernelScene::KernelScene(cui gw, cui gh, cui cs, const float r, sf::Vector2f b1, sf::Vector2f b2, sf::Vector2f g) : 
	gridWidth(gw), gridHeight(gh), cellSize(cs), radius(r), border1(b1), border2(b2), gravity(g)
{

}



void KernelScene::simulate(Particle* grid, int* cellCount, float dt, int substeps)
{
	sf::Clock clock = sf::Clock::Clock();
	sf::Time prev = clock.getElapsedTime();
	sf::Time cur;

	Particle* device_grid = 0;
	int* device_cells = 0;
	int* device_cells2 = 0;
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);

	cudaStatus = hipMalloc((void**)&device_grid, size * sizeof(Particle));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc 1 failed %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&device_cells, gridWidth * gridHeight * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc 2 failed\n");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&device_cells2, gridWidth * gridHeight * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc 3 failed\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(device_grid, grid, size * sizeof(Particle), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy in 1 failed\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(device_cells, cellCount, gridWidth * gridHeight * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy in 2 failed\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(device_cells2, cellCount, gridWidth * gridHeight * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy in 3 failed\n");
		goto Error;
	}

	//std::cout << "\n1: " << std::to_string(clock.getElapsedTime().asSeconds()) << '\n';
	//kernelSimulate <<< NUM_SM * MAX_BLOCKS, 1024>>> (device_grid, device_cells, gridWidth, gridHeight, cellSize, radius, dt, substeps, border1, border2, gravity);
	for (int k{}; k < 1; ++k)
	{
		applyGravityKernel << < NUM_SM * MAX_BLOCKS, 1024 >> > (device_grid, device_cells, gridWidth, gridHeight, cellSize, gravity);
		hipDeviceSynchronize();

		calculateCollisions <<< NUM_SM * MAX_BLOCKS, 1024 >>> (device_grid, device_cells, gridWidth, gridHeight, cellSize, radius, device_cells2);
		hipDeviceSynchronize();	
		applyCollisionsKernel << < NUM_SM * MAX_BLOCKS, 1024 >> > (device_grid, device_cells, gridWidth, gridHeight, cellSize, radius);
		hipDeviceSynchronize();

		applyConstraintsKernel << < NUM_SM * MAX_BLOCKS, 1024 >> > (device_grid, device_cells, gridWidth, gridHeight, cellSize, border1, border2);
		hipDeviceSynchronize();

		calculatePositionsKernel << < NUM_SM * MAX_BLOCKS, 1024 >> > (device_grid, device_cells, gridWidth, gridHeight, cellSize, dt/(float)substeps);	
	}
	
	//std::cout << "\n2: " << std::to_string(clock.getElapsedTime().asSeconds()) << '\n';
	//clock = sf::Clock::Clock();
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
	//std::cout << "3: " << std::to_string(clock.getElapsedTime().asSeconds()) << '\n';
	//clock = sf::Clock::Clock();
	cudaStatus = hipMemcpy(grid, device_grid, size * sizeof(Particle), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy out failed!");
	}
	cudaStatus = hipMemcpy(cellCount, device_cells, gridWidth * gridHeight * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy out failed!");
	}


	//std::cout << "4: " << std::to_string(clock.getElapsedTime().asSeconds()) << '\n';
Error:
	hipFree(device_grid);
	hipFree(device_cells);
	
}