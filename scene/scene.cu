#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""
#include "scene.cuh"
#include <iostream>
#include <stdio.h>
#include <SFML/Graphics.hpp>
#include <chrono>
#include <string>

__device__ void pAtomicCpy(Particle& p1, Particle& p2)
{


	atomicExch(&p1.m_radius, p2.m_radius);

	atomicExch(&p1.m_curPos.x, p2.m_curPos.x);
	atomicExch(&p1.m_curPos.y, p2.m_curPos.y);

	atomicExch(&p1.m_prevPos.x, p2.m_prevPos.x);
	atomicExch(&p1.m_prevPos.y, p2.m_prevPos.y);
	
	atomicExch(&p1.m_pressure, p2.m_pressure);
	atomicExch(&p1.m_pressureCoef, p2.m_pressureCoef);

	//atomicExch(&p1.m_color.r, p2.m_color.r);
	//atomicExch(&p1.m_color.g, p2.m_color.g);
	//atomicExch(&p1.m_color.b, p2.m_color.b);

	atomicExch(&p1.m_acceleration.x, p2.m_acceleration.x);
	atomicExch(&p1.m_acceleration.y, p2.m_acceleration.y);

}

__device__ bool pCmp(Particle& p1, Particle& p2)
{
	return p1.m_curPos.x == p2.m_curPos.x && p1.m_curPos.y == p2.m_curPos.y &&
		p1.m_prevPos.x == p2.m_prevPos.x && p1.m_prevPos.y == p2.m_prevPos.y &&
		p1.m_radius == p2.m_radius;
}
__device__ void pCpy(Particle& p1, Particle& other)
{
	p1.m_curPos = other.m_curPos;
	p1.m_prevPos = other.m_prevPos;
	p1.m_radius = other.m_radius;
	p1.m_pressure = other.m_pressure;
	p1.m_pressureCoef = other.m_pressureCoef;
	p1.m_color = other.m_color;
	p1.m_acceleration = other.m_acceleration;
	p1.returnColorPressure = other.returnColorPressure;
}

__device__ void kernelCalculatePos(Particle& p, const float dt)
{
	const float vel_x = p.m_curPos.x - p.m_prevPos.x, vel_y = p.m_curPos.y - p.m_prevPos.y;
	p.m_prevPos.x = p.m_curPos.x;
	p.m_prevPos.y = p.m_curPos.y;
	const float m = sqrtf(vel_x * vel_x + vel_y * vel_y);
	const float mc = p.m_pressure * p.m_pressure;
	const float pc = 1.f / (1.f + mc);

	p.m_curPos.x += (vel_x + (p.m_acceleration.x * dt * dt)) * pc;
	p.m_curPos.y += (vel_y + (p.m_acceleration.y * dt * dt)) * pc;

	p.m_pressureCoef = pc;
	p.m_pressure = 0.f;
}

__device__ void kernelAccelerate(Particle& p, sf::Vector2f a)
{
	p.m_acceleration.x += a.x;
	p.m_acceleration.y += a.y;
}

__device__ void kernelApplyConstraint(Particle& p, sf::Vector2f pos1, sf::Vector2f pos2)
{
	const float
		x = p.m_curPos.x,
		y = p.m_curPos.y,
		r = p.m_radius;
	if (x + r > pos2.x)
		p.m_curPos.x = pos2.x - r;
	if (y + r > pos2.y)
		p.m_curPos.y = pos2.y - r;
	if (x - r < pos1.x)
		p.m_curPos.x = pos1.x + r;
	if (y - r < pos1.y)
		p.m_curPos.y = pos1.y + r;
}

__device__ void kernelCollide(Particle& p1, Particle& p2)
{
	const float ax = p1.m_curPos.x - p2.m_curPos.x;
	const float ay = p1.m_curPos.y - p2.m_curPos.y;
	const float ndist = (ax * ax) + (ay * ay);
	const float r2 = p1.m_radius + p2.m_radius;
	if (ndist > r2 * r2)
		return;
	const float dist = sqrtf(ndist); 
	if (!dist)
		return;
	const float delta = ((r2 - dist)) * 0.5f;
	const float nx = ax / dist * delta;
	const float ny = ay / dist * delta;;
	p1.m_curPos.x += nx;
	p1.m_curPos.y += ny;
	p2.m_curPos.x -= nx;
	p2.m_curPos.y -= ny;
	p1.m_pressure += delta / 2.f;
	p2.m_pressure += delta / 2.f;
	
}



__device__ void staticGridCollide(cui pos1, cui pos2, int* grid, int* cellCount, Particle* particles, cui gridWidth, cui gridHeight, cui cellSize)
{
	if (pos2 < 0 || pos2 >= gridWidth * gridHeight)
		return;

	for (int i{}; i < cellCount[pos1]; ++i)
		for (int j{}; j < cellCount[pos2]; ++j)
		{
			kernelCollide(particles[grid[pos1 * cellSize + i]], particles[grid[pos2 * cellSize + j]]);
		}
	
}

__global__ void calculateCollisions(int* grid, int* cellCount, Particle* particles, cui gridWidth, cui gridHeight, cui cellSize, float radius, cui count)
{
	const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= count) return;
	// �������� ����������
	int x = (int)(particles[i].m_curPos.x / (radius * 2.f));
	int y = (int)(particles[i].m_curPos.y / (radius * 2.f));
	// �������� ���������
	if (x < 0)
		x = 0;
	if (y < 0)
		y = 0;
	if (x >= gridWidth)
		x = gridWidth - 1;
	if (y >= gridHeight)
		y = gridHeight - 1;
	int pos2 = x * gridHeight + y;
	if (pos2 != i && cellCount[pos2] < cellSize)
	{
		int old = atomicAdd(&cellCount[pos2], 1);
		grid[pos2 * cellSize + old] = i;
	}
}

__global__ void applyGravityKernel(Particle* particles, cui count, sf::Vector2f g)
{
	const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= count) return;
	kernelAccelerate(particles[i], g);
}

__global__ void applyCollisionsKernel(int* grid, int* cellCount, Particle* particles, cui gridWidth, cui gridHeight, cui cellSize, float radius)
{
	const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= gridHeight * gridWidth) return;
	for (int j{}; j < cellCount[i]; ++j)
	{
		staticGridCollide(i, i, grid, cellCount, particles, gridWidth, gridHeight, cellSize);
		staticGridCollide(i, i + 1, grid, cellCount, particles, gridWidth, gridHeight, cellSize);
		staticGridCollide(i, i - 1, grid, cellCount, particles, gridWidth, gridHeight, cellSize);
		staticGridCollide(i, i + gridHeight, grid, cellCount, particles, gridWidth, gridHeight, cellSize);
		staticGridCollide(i, i + gridHeight + 1, grid, cellCount, particles, gridWidth, gridHeight, cellSize);
		staticGridCollide(i, i + gridHeight - 1, grid, cellCount, particles, gridWidth, gridHeight, cellSize);
		staticGridCollide(i, i - gridHeight, grid, cellCount, particles, gridWidth, gridHeight, cellSize);
		staticGridCollide(i, i - gridHeight + 1, grid, cellCount, particles, gridWidth, gridHeight, cellSize);
		staticGridCollide(i, i - gridHeight - 1, grid, cellCount, particles, gridWidth, gridHeight, cellSize);
	}
}

__global__ void applyConstraintsKernel(Particle* particles, cui count, sf::Vector2f b1, sf::Vector2f b2)
{
	const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= count) return;
	kernelApplyConstraint(particles[i], b1, b2);
}

__global__ void calculatePositionsKernel(Particle* particles, cui count, float dt)
{
	const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= count) return;
	kernelCalculatePos(particles[i], dt);
}

__global__ void emptyGrid(int* grid, int* cellCount, cui gridWidth, cui gridHeight, cui cellSize)
{
	const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= gridHeight * gridWidth) return;
	cellCount[i] = 0;
}

KernelScene::KernelScene(cui gw, cui gh, cui cs, const float r, sf::Vector2f b1, sf::Vector2f b2, sf::Vector2f g) : 
	gridWidth(gw), gridHeight(gh), cellSize(cs), radius(r), border1(b1), border2(b2), gravity(g)
{

}



void KernelScene::simulate(Particle* p, int count, float dt, int substeps)
{
	sf::Clock clock = sf::Clock::Clock();
	sf::Time prev = clock.getElapsedTime();
	sf::Time cur;

	Particle* device_particles;

	int* device_grid = 0;
	int* device_cells = 0;

	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);

	cudaStatus = hipMalloc((void**)&device_grid, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc 1 failed %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&device_cells, gridWidth * gridHeight * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc 2 failed\n");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&device_particles, count * sizeof(Particle));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc 3 failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipMemcpy(device_particles, p, count * sizeof(Particle), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy in 3 failed\n");
		goto Error;
	}

	for (int k{}; k < substeps; ++k)
	{
		emptyGrid << < NUM_SM * MAX_BLOCKS, 1024 >> > (device_grid, device_cells, gridWidth, gridHeight, cellSize);

		calculateCollisions <<< NUM_SM * MAX_BLOCKS, 1024 >>> (device_grid, device_cells, device_particles, gridWidth, gridHeight, cellSize, radius, count);

		applyGravityKernel << < NUM_SM * MAX_BLOCKS, 1024 >> > (device_particles, count, gravity);


		applyCollisionsKernel << < NUM_SM * MAX_BLOCKS, 1024 >> > (device_grid, device_cells, device_particles,gridWidth, gridHeight, cellSize, radius);

		applyConstraintsKernel << < NUM_SM * MAX_BLOCKS, 1024 >> > (device_particles, count, border1, border2);

		calculatePositionsKernel << < NUM_SM * MAX_BLOCKS, 1024 >> > (device_particles, count, dt/(float)substeps);
	}


	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
	
	cudaStatus = hipMemcpy(p, device_particles, count * sizeof(Particle), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy out failed\n %s", hipGetErrorString(cudaStatus));
	}
	

Error:
	hipFree(device_grid);
	hipFree(device_cells);
	
}